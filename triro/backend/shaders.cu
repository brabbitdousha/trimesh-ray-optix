#include "hip/hip_runtime.h"

#include "LaunchParams.h"
#include "optix_types.h"
#include <cuda_device_runtime_api.h>
#include <optix_device.h>
#include <tuple>

namespace hmesh {

extern "C" __constant__ LaunchParams launchParams;

__forceinline__ __device__ std::tuple<unsigned int, unsigned int>
setPayloadPointer(void *p) {
    unsigned int u0 = (unsigned long long)p & 0xFFFFFFFFllu;
    unsigned int u1 = ((unsigned long long)p >> 32) & 0xFFFFFFFFllu;
    return {u0, u1};
}

template <typename T>
__forceinline__ __host__ __device__ T *getPayloadPointer() {
    unsigned int u0 = optixGetPayload_0();
    unsigned int u1 = optixGetPayload_1();
    void *p = (void *)(((unsigned long long)u1 << 32) + u0);
    return (T *)p;
}

// intersects_any

extern "C" __global__ void __miss__intersectsAny() {
    bool *result_pt = getPayloadPointer<bool>();
    *result_pt = false;
}

extern "C" __global__ void __anyhit__intersectsAny() {
    bool *result_pt = getPayloadPointer<bool>();
    *result_pt = true;
}

extern "C" __global__ void __raygen__intersectsAny() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    // intersection result, to be overwritten by the shader
    bool isect_result = false;
    // ray info
    float3 ray_origin = launchParams.rays.origins[idx];
    float3 ray_dir = launchParams.rays.directions[idx];
    // result pointer
    auto [u0, u1] = setPayloadPointer(&isect_result);
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 0., 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 0, 0, u0, u1);
    launchParams.results.hit[idx] = isect_result;
}

// intersects_first

extern "C" __global__ void __miss__intersectsFirst() {
    int *result_pt = getPayloadPointer<int>();
    *result_pt = -1;
}

extern "C" __global__ void __closesthit__intersectsFirst() {
    int *result_pt = getPayloadPointer<int>();
    *result_pt = optixGetPrimitiveIndex();
}

extern "C" __global__ void __raygen__intersectsFirst() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    // first hit triangle index, to be overwritten by the shader
    int ch_idx = -1;
    // ray info
    float3 ray_origin = launchParams.rays.origins[idx];
    float3 ray_dir = launchParams.rays.directions[idx];
    // result pointer
    auto [u0, u1] = setPayloadPointer(&ch_idx);
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 0., 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 0, 0,
               u0, u1);
    launchParams.results.triIdx[idx] = ch_idx;
}

// intersects_closest

struct WBData {
    bool hit;
    bool front;
    int triIdx;
    float3 loc;
    float2 uv;
};

extern "C" __global__ void __miss__intersectsClosest() {
    WBData *result = getPayloadPointer<WBData>();
    result->hit = false;
    result->triIdx = -1;
    result->uv = {0, 0};
    result->loc = {0, 0, 0};
    result->front = false;
}

extern "C" __global__ void __closesthit__intersectsClosest() {
    WBData *result = getPayloadPointer<WBData>();
    float2 uv = optixGetTriangleBarycentrics();
    int triIdx = optixGetPrimitiveIndex();
    float3 verts[3];
    optixGetTriangleVertexData(launchParams.traversable, triIdx, 0, 0, verts);
    float3 isectLoc = {
        uv.x * verts[1].x + uv.y * verts[2].x + (1 - uv.x - uv.y) * verts[0].x,
        uv.x * verts[1].y + uv.y * verts[2].y + (1 - uv.x - uv.y) * verts[0].y,
        uv.x * verts[1].z + uv.y * verts[2].z + (1 - uv.x - uv.y) * verts[0].z};

    result->triIdx = triIdx;
    result->uv = uv;
    result->hit = true;
    result->front = optixIsFrontFaceHit();
    result->loc = isectLoc;
}

extern "C" __global__ void __raygen__intersectsClosest() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    WBData wbdata;
    // ray info
    float3 ray_origin = launchParams.rays.origins[idx];
    float3 ray_dir = launchParams.rays.directions[idx];
    // result pointer
    auto [u0, u1] = setPayloadPointer(&wbdata);
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 0., 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 0, 0,
               u0, u1);
    // write back to the buffers
    launchParams.results.hit[idx] = wbdata.hit;
    launchParams.results.front[idx] = wbdata.front;
    launchParams.results.location[idx] = wbdata.loc;
    launchParams.results.triIdx[idx] = wbdata.triIdx;
    launchParams.results.uv[idx] = wbdata.uv;
}

// intersects_location

extern "C" __global__ void __anyhit__intersectsCount() {
    int *hitCount = getPayloadPointer<int>();
    // it seems we don't need atomic ops as they are not parallel
    (*hitCount)++;
    optixIgnoreIntersection();
}

extern "C" __global__ void __raygen__intersectsCount() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    int hitCount = 0;
    // ray info
    float3 ray_origin = launchParams.rays.origins[idx];
    float3 ray_dir = launchParams.rays.directions[idx];
    // result pointer
    auto [u0, u1] = setPayloadPointer(&hitCount);
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 0., 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 0, 0, u0, u1);
    launchParams.results.hitCount[idx] = hitCount;
}

struct IsectLocWBTerm {
    int triIdx;
    float3 loc;
}; // 16B

struct IsectLocPayload {
    IsectLocWBTerm terms[MAX_ANYHIT_SIZE];
    int hitCount;
    int globalIdx;
};

extern "C" __global__ void __anyhit__intersectsLocation() {
    IsectLocPayload *payload = getPayloadPointer<IsectLocPayload>();
    if (payload->hitCount >= MAX_ANYHIT_SIZE)
        return;
    int localidx = payload->hitCount;
    payload->hitCount++;
    float2 uv = optixGetTriangleBarycentrics();
    int triIdx = optixGetPrimitiveIndex();
    float3 verts[3];
    optixGetTriangleVertexData(launchParams.traversable, triIdx, 0, 0, verts);
    float3 isectLoc = {
        uv.x * verts[1].x + uv.y * verts[2].x + (1 - uv.x - uv.y) * verts[0].x,
        uv.x * verts[1].y + uv.y * verts[2].y + (1 - uv.x - uv.y) * verts[0].y,
        uv.x * verts[1].z + uv.y * verts[2].z + (1 - uv.x - uv.y) * verts[0].z};
    payload->terms[localidx].loc = isectLoc;
    payload->terms[localidx].triIdx = triIdx;
    optixIgnoreIntersection();
}

extern "C" __global__ void __raygen__intersectsLocation() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    int hitCount = launchParams.rays.hitCounts[idx];
    int globalIdx = launchParams.rays.globalIdx[idx];
    IsectLocPayload payload = {};
    payload.hitCount = 0;
    payload.globalIdx = globalIdx;
    // ray info
    float3 ray_origin = launchParams.rays.origins[idx];
    float3 ray_dir = launchParams.rays.directions[idx];
    // result pointer
    auto [u0, u1] = setPayloadPointer(&payload);
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 0., 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 0, 0, u0, u1);
    // fill global buffer
    printf("idx: %d, hitcount: %d, globalIdx: %d\n", idx, hitCount, globalIdx);
    for (int i = 0; i < hitCount; i++) {
        launchParams.results.rayIdx[globalIdx + i] = idx;
        launchParams.results.triIdx[globalIdx + i] = payload.terms[i].triIdx;
        launchParams.results.location[globalIdx + i] = payload.terms[i].loc;
    }
}

} // namespace hmesh
