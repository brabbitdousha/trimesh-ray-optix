#include "hip/hip_runtime.h"

#include "LaunchParams.h"
#include "optix_types.h"
#include <optix_device.h>

namespace hmesh {

extern "C" __constant__ LaunchParams launchParams;

// intersects_any

extern "C" __global__ void __miss__intersectsAny() {
    unsigned long long ptvalue =
        ((unsigned long long)optixGetPayload_1() << 32) + optixGetPayload_0();
    bool *result_pt = (bool *)ptvalue;
    *result_pt = false;
}

extern "C" __global__ void __closesthit__intersectsAny() {
    unsigned long long ptvalue =
        ((unsigned long long)optixGetPayload_1() << 32) + optixGetPayload_0();
    bool *result_pt = (bool *)ptvalue;
    *result_pt = true;
}

extern "C" __global__ void __raygen__intersectsAny() {
    // thread index, ranging in [0, N)
    int idx = optixGetLaunchIndex().x;
    // intersection result, to be overwritten by the shader
    bool isect_result = false;
    // ray info
    float3 ray_origin =
        *(float3 *)(launchParams.origins + idx * 3);
    float3 ray_dir = *(float3 *)(launchParams.dirs + idx * 3);
    // result pointer
    unsigned int u0 = (unsigned long long)(&isect_result) & 0xFFFFFFFFllu;
    unsigned int u1 =
        ((unsigned long long)(&isect_result) >> 32) & 0xFFFFFFFFllu;
    optixTrace(launchParams.traversable, ray_origin, ray_dir, 1e-4, 1e7, 0,
               OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 0, 0, u0, u1);
    launchParams.result[idx] = isect_result;
    printf("ray origin: %f, %f, %f, ray dir: %f, %f, %f, idx: %i\n",
           ray_origin.x, ray_origin.y, ray_origin.z,
           ray_dir.x, ray_dir.y, ray_dir.z, idx);
}

} // namespace hmesh
